#include <iostream>
#include <hip/hip_runtime.h>

#define N 1000000  // Number of elements in the vectors

__global__ void vectorAdd(const float *A, const float *B, float *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    float *h_A, *h_B, *h_C;  // Host pointers
    float *d_A, *d_B, *d_C;  // Device pointers

    // Allocate host memory
    h_A = (float*)malloc(N * sizeof(float));
    h_B = (float*)malloc(N * sizeof(float));
    h_C = (float*)malloc(N * sizeof(float));

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i);
    }

    // Allocate device memory
    checkCudaErrors(hipMalloc((void**)&d_A, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&d_B, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&d_C, N * sizeof(float)));

    // Copy vectors from host to device
    checkCudaErrors(hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice));

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);
    hipEventRecord(stop);

    // Copy result back to host
    checkCudaErrors(hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

    // Wait for the GPU to finish
    hipEventSynchronize(stop);

    // Measure elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time taken for vector addition: " << milliseconds << " ms" << std::endl;

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
